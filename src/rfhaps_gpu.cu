#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include "arrayDeleter.h"
#include "rfhaps_gpu.h"
#include <algorithm>
#include "R.h"
#include <exception>
#include "getFunnelGPU.h"
using namespace std;

#define SAFE_EXIT( m )\
  Rprintf("%s in file '%s' in line %i.\n", m,  __FILE__, __LINE__);\
  hipDeviceReset();\
  exit(EXIT_FAILURE);

#define R_CUDA_SAFE_CALL( call )\
  {\
  hipError_t hipError_t = call ;\
  if( hipError_t != hipSuccess ) {\
    Rprintf("%s in file '%s' in line %i.\n", hipGetErrorString(hipError_t),  __FILE__, __LINE__);\
    hipDeviceReset();\
    exit(EXIT_FAILURE);\
  }\
}

void selectGPU(int deviceNum) {
  int myDevice, numDevices;

  R_CUDA_SAFE_CALL( hipGetDeviceCount( &numDevices ) );
  if (deviceNum > numDevices) {
    Rprintf("Unable to use device %i, only %i found.\n",deviceNum,numDevices);
  }
  if (deviceNum >= 0) {   
    // if caller specified a device then use it
    R_CUDA_SAFE_CALL( hipSetDevice(deviceNum) );
  } else if (deviceNum == -1) {
    // take the first available (will share unless GPUs are in exclusive mode)
    Rprintf("Selecting first available GPU.\n");
    R_CUDA_SAFE_CALL( hipSetDevice(0) );
  } else if (deviceNum == -2) {
    // try some smarts to round robin assign devices based on the MPI local rank
    char* cLocalRank;
    int localRank = 0;
    cLocalRank = getenv("OMPI_COMM_WORLD_LOCAL_RANK");
    if (cLocalRank!=NULL) {
      localRank = atoi(cLocalRank);
      Rprintf("Local rank is: %i.\n",localRank);
    } else {
      Rprintf("Unable to determine local rank.\n");
    }
    R_CUDA_SAFE_CALL( hipSetDevice(localRank % numDevices) );
  } else {
    SAFE_EXIT("Unknown argument to selectGPU");
  }

  // which device did we end up with..
  R_CUDA_SAFE_CALL( hipGetDevice( &myDevice ) );
  Rprintf("Using device %i.\n",myDevice);
}


template<int nFounders> __device__ void pr2pt(double r, double *prob);
template<> __device__ void pr2pt<4>(double r, double *prob)
{
	prob[0] = (1-r)/(4+8*r);
	prob[1] = r/(4+8*r);
	prob[2] = r/(4+8*r);
}
template<> __device__ void pr2pt<8>(double r, double *prob)
{
	prob[0] = (1-r)*(1-r)/(8+16*r);
	prob[1] = r*(1-r)/(8+16*r);
	prob[2] = r/(16+32*r);
}
template<int nFounders>__device__ void pr2ptirip(double r, int s, double *prob);
template<> __device__ void pr2ptirip<4>(double r, int s, double *prob)
{
  prob[0]=(pow(1-r, 2+s-1)/4+(2*r+1-pow(1-r, s-1))/16)/(1+2*r); 
  prob[1]=prob[2]=(1-4*prob[0])/12;
}
template<> __device__ void pr2ptirip<8>(double r, int s, double* prob)
{
        double tmp = pow(1-r, s-1);
        prob[0] = (tmp *(1-r)*(1-r)*(1-r)/8 + (2*r + 1 - tmp)/64)/(1 + 2*r);
        prob[1] = prob[2] = (1 - 8 * prob[0]) / 56;
}
extern __shared__ char dyn_shared_mem[]; /* dynamic allocated shared memory */

template<int nFounders>
__global__ void gpu_rfhaps(int nRecomb, int* ngen, 
			   int nPairs, int nFinals,
	   		   int *finalg, int* pair1, int* pair2, 
			   double *thvec, int* markerPatternIDs, bool* allowableMarkerPatterns, int nMarkerPatterns, double* lineWeights, double* output) {
        /*
	 *	Mask is a matrix that looks something like
	 *	   nfounders = 8      	nfounders = 4
	 *	   01222222		0122
 	 *	   10222222		1022
 	 *	   22012222		2201
 	 *	   22102222		2210
 	 *	   22220122
 	 *	   22221022
 	 *	   22222201
 	 *	   22222210
 	 */
	__shared__ int mask[8][8];
	int g1[8];
	int g2[8];
	/* I suppose this could be done by differently..
	 * by having each thread copy a portion of the mask from device memory
	 * seems pointless unless the mask gets very large
	 * I assume doing it this way just increases the code size
	 */
	mask[0][0]=0;	mask[0][1]=1;	mask[0][2]=2;	mask[0][3]=2;	mask[0][4]=2;	mask[0][5]=2;	mask[0][6]=2;	mask[0][7]=2;
	mask[1][0]=1;	mask[1][1]=0;	mask[1][2]=2;	mask[1][3]=2;	mask[1][4]=2;	mask[1][5]=2;	mask[1][6]=2;	mask[1][7]=2;
	mask[2][0]=2;	mask[2][1]=2;	mask[2][2]=0;	mask[2][3]=1;	mask[2][4]=2;	mask[2][5]=2;	mask[2][6]=2;	mask[2][7]=2;
	mask[3][0]=2;	mask[3][1]=2;	mask[3][2]=1;	mask[3][3]=0;	mask[3][4]=2;	mask[3][5]=2;	mask[3][6]=2;	mask[3][7]=2;
	mask[4][0]=2;	mask[4][1]=2;	mask[4][2]=2;	mask[4][3]=2;	mask[4][4]=0;	mask[4][5]=1;	mask[4][6]=2;	mask[4][7]=2;
	mask[5][0]=2;	mask[5][1]=2;	mask[5][2]=2;	mask[5][3]=2;	mask[5][4]=1;	mask[5][5]=0;	mask[5][6]=2;	mask[5][7]=2;
	mask[6][0]=2;	mask[6][1]=2;	mask[6][2]=2;	mask[6][3]=2;	mask[6][4]=2;	mask[6][5]=2;	mask[6][6]=0;	mask[6][7]=1;
	mask[7][0]=2;	mask[7][1]=2;	mask[7][2]=2;	mask[7][3]=2;	mask[7][4]=2;	mask[7][5]=2;	mask[7][6]=1;	mask[7][7]=0;
	double *shm_thvec = (double*)dyn_shared_mem; /* dynamically allocated shared memory */
	shm_thvec[threadIdx.x] = thvec[threadIdx.x];
	__syncthreads();

	/* work out which part of the r and k loops in the CPU implementation
	 * we are responsible for
	 */
	int r = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;
	if (k>=nPairs) return;
	for (int i = 0; i < nFinals; i++) {
//	  assert(k >= 0 && i >= 0);
//	  assert(k < npairs && i < nfinals);
	  
	  /* TODO something seems to be corrupt in the finalg data
	   * need to find the cause of these rogue indiv values */

	  int p1 = pair1[k];
	  int p2 = pair2[k];
//	  assert(p1 >= 0 && p2 >= 0);
//	  assert(p1 < nmrk);
//	  assert(p2 < nmrk);

	  for (int j=0; j<nFounders; j++)
		g1[j] = g2[j] = 0;
	  
	  /* point to start of genotypes for the individual */
	  int h1 = finalg[p1*nFinals+i];
	  int h2 = finalg[p2*nFinals+i];
	  if ((h1>0)*(h2>0)) {  	/* check for missing values */
	    double theta = shm_thvec[r];
	    double probclass[3];
	    
	    if ((h1&1) == 1) {g1[0]=1; h1 -= 1; }
	    if ((h1&3) == 2) {g1[1]=1; h1 -= 2; }
	    if ((h1&7) == 4) {g1[2]=1; h1 -= 4; }
	    if ((h1&15) == 8){g1[3]=1; h1 -= 8; }
	    if ((h1&31) == 16){g1[4]=1; h1 -= 16;}
	    if ((h1&63) == 32){g1[5]=1; h1 -= 32;}
	    if ((h1&127) == 64){g1[6]=1; h1 -= 64;}
	    if ((h1&255) == 128){g1[7]=1;}

	    if ((h2&1) == 1) {g2[0]=1; h2 -= 1; }
	    if ((h2&3) == 2) {g2[1]=1; h2 -= 2; }
	    if ((h2&7) == 4) {g2[2]=1; h2 -= 4; }
	    if ((h2&15) == 8){g2[3]=1; h2 -= 8; }
	    if ((h2&31) == 16){g2[4]=1; h2 -= 16;}
	    if ((h2&63) == 32){g2[5]=1; h2 -= 32;}
	    if ((h2&127) == 64){g2[6]=1; h2 -= 64;}
	    if ((h2&255) == 128){g2[7]=1;}

	    /* Compute haplotype probabilities based on theta */
	    /* TODO template gpu_rfhaps based on ngen
	     * this will allow us to use 7 fewer registers
	     * for the case when ngen == 0
	     * and 2 fewer when ngen > 0
	     * it adds annoying complexity to the kernel invocation call portion though
	     */
		if(ngen[i] == 0)
		{
			pr2pt<nFounders>(theta, probclass);
		}
		else
		{
			pr2ptirip<nFounders>(theta, ngen[i], probclass);
		}
	    /* Check whether progeny genotypes are compatible with parent genotypes */
	    
	    /* For each combination of haplotypes which is feasible
	     * add the haplotype probabilities together */
	    double hp = 0;
	    for (int j=0; j<nFounders; j++){
	      for (int l=0; l<nFounders; l++){
		if (g1[j]*g2[l]) {
		  hp += probclass[mask[j][l]];
		}
	      }
	    }
	    /* log10(hp) is the individual contribution to the log-likelihood */
	    output[k*nRecomb+r] += (allowableMarkerPatterns[nMarkerPatterns * markerPatternIDs[p1] + markerPatternIDs[p2]] ? lineWeights[i]*log10(hp) : 0);
	    
	  } // end of check for missing values
  }
}
struct rfhaps_gpu_internal_args
{
	int* pair1, *pair2;
	long pairsOffset, nPairsToCalculate;
	int nMarkers, nFinals, recombOffset, nRecombToCalculate, nRecomb, nFounders;
	int* finalsD, *nIntercrossingD;
	double* recombinationFractionsD;
	bool hasAI;
	double* output;
	int marker2RangeSize, marker1Start, marker2Start;
	int* markerPatternIDs;
	bool* allowableMarkerPatterns;
	int nMarkerPatterns;
	double* lineWeightsD;
};
pedigreeColumns::pedigreeColumns(int* id, int* Male, int* Female, int* Observed, std::vector<std::string>& Design)
: id(id), Male(Male), Female(Female), Observed(Observed), Design(Design)
{}
__host__ bool rfhaps_gpu_internal(rfhaps_gpu_internal_args& args)
{
	assert(args.nFounders == 4 || args.nFounders == 8);
	const long resultSize = args.nPairsToCalculate * args.nRecombToCalculate;

	double* outputD;
	hipMalloc((void**)&outputD, resultSize*sizeof(double));
	//hipMemcpy(outputD, args.output, sizeof(double)*resultSize, hipMemcpyHostToDevice);
	hipMemset(outputD, 0, sizeof(double)*resultSize);


	//transfer pairs data
	int* pair1D, *pair2D;
	hipMalloc((int**)&pair1D, args.nPairsToCalculate * sizeof(int));
	hipMemcpy(pair1D, args.pair1 + args.pairsOffset, args.nPairsToCalculate*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((int**)&pair2D, args.nPairsToCalculate * sizeof(int));
	hipMemcpy(pair2D, args.pair2 + args.pairsOffset, args.nPairsToCalculate*sizeof(int), hipMemcpyHostToDevice);

	int threadsX = args.nRecombToCalculate;
	int threadsY = args.nPairsToCalculate;

	dim3 dimBlock(threadsX, floor(440/threadsX)); // logical max of 512 threads per block (only 440 per multiprocessor on Fermi anyway)
	dim3 dimGrid(1, ceil((double)threadsY / (double)dimBlock.y));
	size_t dynSharedSize = sizeof(double)*args.nRecombToCalculate;

	if (args.nFounders==4) 
	{
		gpu_rfhaps<4><<<dimGrid, dimBlock, dynSharedSize>>>(args.nRecombToCalculate, args.nIntercrossingD, args.nPairsToCalculate, args.nFinals, args.finalsD, pair1D, pair2D, args.recombinationFractionsD + args.recombOffset, args.markerPatternIDs, args.allowableMarkerPatterns, args.nMarkerPatterns, args.lineWeightsD, outputD);
	}
	else if (args.nFounders==8)
	{
		gpu_rfhaps<8><<<dimGrid, dimBlock, dynSharedSize>>>(args.nRecombToCalculate, args.nIntercrossingD, args.nPairsToCalculate, args.nFinals, args.finalsD, pair1D, pair2D, args.recombinationFractionsD + args.recombOffset, args.markerPatternIDs, args.allowableMarkerPatterns, args.nMarkerPatterns, args.lineWeightsD, outputD);
	}
	else
	{
		Rprintf("nFounders must have value 4 or 8\n");
		exit(-1);
	}
	bool result = true;
	hipDeviceSynchronize();
	hipError_t lastError = hipGetLastError();
	if(lastError != hipSuccess)
	{
	  Rprintf("CUDA Last Error: %s\n",hipGetErrorString(lastError));
		result = false;
	}

	double* copiedOutput = new double[resultSize];;
	hipMemcpy(copiedOutput, outputD, resultSize*sizeof(double), hipMemcpyDeviceToHost);

	for(int pairCounter = 0; pairCounter < args.nPairsToCalculate; pairCounter++)
	{
		int markerCounter2 = args.pair2[pairCounter + args.pairsOffset];
		int markerCounter1 = args.pair1[pairCounter + args.pairsOffset];
		for(int recombCounter = 0; recombCounter < args.nRecombToCalculate; recombCounter++)
		{
			//Turns out this overflows the range of a signed int
			long index = (long)(markerCounter1 - args.marker1Start)*(long)args.nRecomb*(long)args.marker2RangeSize + (long)(markerCounter2-args.marker2Start)*(long)args.nRecomb + (long)(recombCounter+args.recombOffset);
			args.output[index] += copiedOutput[(long)pairCounter*(long)args.nRecombToCalculate + (long)recombCounter];
		}
	}
	hipFree(outputD);
	hipFree(pair1D);
	hipFree(pair2D);
	delete[] copiedOutput;
	return result;
}
extern "C" __host__ bool rfhaps_gpu(rfhaps_gpu_args& args)
{
        selectGPU(args.deviceNum);

	int marker2RangeSize = args.marker2End - args.marker2Start, marker1RangeSize = args.marker1End - args.marker1Start;
	int nMarkers = args.markerPatternIDs.size();
	int nMarkerPatterns = args.markerEncodings.size();
	const int finalsSize = nMarkers * args.nFinals;
	int* copiedFinals = new int[finalsSize];

	assert(args.nFounders == 4 || args.nFounders == 8);

	//working out the number of pairs is complicated because if we have a region on the diagonal we use the symmetry to avoid making double calculations. Whereas if we have a bit on the 
	//off-diagonal we need to calculate every value
	int maxStart = std::max(args.marker1Start, args.marker2Start);
	int minEnd = std::min(args.marker1End, args.marker2End);
	long square = std::max(minEnd - maxStart, 0);
	long squarePairs = square*(square + 1) /2;
	long nPairs = (marker1RangeSize * marker2RangeSize) - square * square + squarePairs;

	//re-encode finals genetic data, so that now the 1st bit says whether that individual is compatible with founder 1, 2nd bit compatible with founder 2, etc
	intArray8 funnel_;
	for(int individualCounter = 0; individualCounter < args.nFinals; individualCounter++)
	{
		funnel_ = args.funnels[individualCounter];
		for(int markerCounter = 0; markerCounter < nMarkers; markerCounter++)
		{
			int newValue = 0;
			int oldValue = args.finals[individualCounter+args.nFinals*markerCounter];
			for(int founderCounter = 0; founderCounter < args.nFounders; founderCounter++)
			{
				if(oldValue == args.founders[funnel_.val[founderCounter] - 1 + args.nFounders*markerCounter]) newValue += (1 << founderCounter);
			}
			copiedFinals[individualCounter+args.nFinals*markerCounter] = newValue;
		}
	}
	//transfer intercrossing data
	int* nIntercrossingD;
	hipMalloc((void**)&nIntercrossingD, args.nFinals * sizeof(int));
	hipMemcpy(nIntercrossingD, args.nIntercrossing, args.nFinals * sizeof(int), hipMemcpyHostToDevice);

	int* pair1 = new int[nPairs], *pair2 = new int[nPairs];
	int* p1Ptr = pair1, *p2Ptr = pair2;
	//generate pairs
	for(int i = args.marker1Start; i < args.marker1End; i++)
	{
		for(int j = args.marker2Start; j < args.marker2End; j++)
		{
			if(i >= maxStart && i < minEnd && j >= maxStart && j < minEnd && j < i) continue;
			*p2Ptr = j;
			*p1Ptr = i;
			p1Ptr++; p2Ptr++;
		}
	}

	//copy across final genetic data
	int* finalsD;
	hipMalloc((void**)&finalsD, finalsSize * sizeof(int));
	hipMemcpy(finalsD, copiedFinals, finalsSize * sizeof(int), hipMemcpyHostToDevice);

	delete[] copiedFinals;
	//copy across recombination fractions
	double* recombinationFractionsD;
	hipMalloc((void**)&recombinationFractionsD, args.nRecomb * sizeof(double));
	hipMemcpy(recombinationFractionsD, args.recombination, args.nRecomb * sizeof(double), hipMemcpyHostToDevice);

	//copy across the allowable marker patterns data
	bool* allowableMarkerPatternsD;
	hipMalloc((void**)&allowableMarkerPatternsD, nMarkerPatterns * nMarkerPatterns * sizeof(bool));
	hipMemcpy(allowableMarkerPatternsD, args.allowableMarkerPatterns, nMarkerPatterns * nMarkerPatterns * sizeof(bool), hipMemcpyHostToDevice);
	
	int* markerPatternIDsD;
	hipMalloc((void**)&markerPatternIDsD, args.markerPatternIDs.size() * sizeof(int));
	hipMemcpy(markerPatternIDsD, &(args.markerPatternIDs[0]), args.markerPatternIDs.size() * sizeof(int), hipMemcpyHostToDevice);
	
	//copy across line weights data
	double* lineWeightsD;
	hipMalloc((void**)&lineWeightsD, args.lineWeights.size() * sizeof(double));
	hipMemcpy(lineWeightsD, &(args.lineWeights[0]), args.lineWeights.size() * sizeof(double), hipMemcpyHostToDevice);
	
	int threadsX = args.nRecomb;
	int threadsY = nPairs;

	dim3 dimBlock(threadsX, floor(440/threadsX)); // logical max of 512 threads per block (only 440 per multiprocessor on Fermi anyway)
	dim3 dimGrid(1, ceil((double)threadsY / (double)dimBlock.y));

	int donePairs = 0;
	long pairsPerCall = nPairs;
	if(dimGrid.y > 65535)
	{
		pairsPerCall = 65535 * dimBlock.y;
		dimGrid.y = pairsPerCall/dimBlock.y;
		threadsY = pairsPerCall;
		Rprintf("Splitting into %ld cuda calls....\n", (long)((nPairs+pairsPerCall-1)/pairsPerCall));
	}
	int requiredThreads = threadsX*threadsY;
	int totalThreads = dimBlock.x*dimGrid.x*dimBlock.y*dimGrid.y;
	Rprintf("Total threads needed = %d\n",requiredThreads);
	Rprintf("Threads in grid = %d\n",totalThreads);
	Rprintf("Surplus threads = %d\n\n", totalThreads - requiredThreads); /* these will need to just sit idle */

	Rprintf("Threads per block  %d x %d = %d\n",dimBlock.x,dimBlock.y,dimBlock.x*dimBlock.y);
	Rprintf("Blocks in grid  %d x %d = %d\n",dimGrid.x,dimGrid.y,dimGrid.x*dimGrid.y);
	//END_DEBUG
	rfhaps_gpu_internal_args internal_args;
	internal_args.pair1 = pair1;
	internal_args.pair2 = pair2;
	internal_args.nMarkers = nMarkers;
	internal_args.nFinals = args.nFinals;
	internal_args.recombOffset = 0;
	internal_args.nRecombToCalculate = args.nRecomb;
	internal_args.nRecomb = args.nRecomb;
	internal_args.finalsD = finalsD;
	internal_args.nIntercrossingD = nIntercrossingD;
	internal_args.recombinationFractionsD = recombinationFractionsD;
	internal_args.hasAI = args.hasAI;
	internal_args.nFounders = args.nFounders;
	internal_args.output = args.output;
	internal_args.marker2RangeSize = marker2RangeSize;
	internal_args.marker1Start = args.marker1Start;
	internal_args.marker2Start = args.marker2Start;
	internal_args.markerPatternIDs = markerPatternIDsD;
	internal_args.allowableMarkerPatterns = allowableMarkerPatternsD;
	internal_args.nMarkerPatterns = nMarkerPatterns;
	internal_args.lineWeightsD = lineWeightsD;
	
	int counter = 0;
	while(donePairs < nPairs)
	{
		Rprintf("Making cuda call %d\n", counter+1);
		internal_args.pairsOffset = donePairs;
		if(donePairs + pairsPerCall >= nPairs)
		{
			internal_args.nPairsToCalculate = nPairs - donePairs;
		}
		else internal_args.nPairsToCalculate = pairsPerCall;
		bool result = rfhaps_gpu_internal(internal_args);
		if(!result)
		{
			Rprintf("A CUDA call failed, exiting...\n");
			return false;
		}
		donePairs += internal_args.nPairsToCalculate;
		counter++;
	}

	delete[] pair1;
	delete[] pair2;

	Rprintf("Finished all CUDA calls\n");
	hipFree(finalsD);
	hipFree(lineWeightsD);
	hipFree(nIntercrossingD);
	hipFree(recombinationFractionsD);
	return true;
}

